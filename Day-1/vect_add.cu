#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float* input1, const float* input2, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = input1[idx] + input2[idx];
    }
}

int main() {
    const int size = 10;
    float hostInput1[size], hostInput2[size], hostOutput[size];

    // Initialize input arrays
    for (int i = 0; i < size; ++i) {
        hostInput1[i] = i * 1.0f;
        hostInput2[i] = (size - i) * 1.0f;
    }

    // Device pointers
    float *deviceInput1, *deviceInput2, *deviceOutput;
    hipMalloc(&deviceInput1, size * sizeof(float));
    hipMalloc(&deviceInput2, size * sizeof(float));
    hipMalloc(&deviceOutput, size * sizeof(float));

    // Copy data from host to device
    hipMemcpy(deviceInput1, hostInput1, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    vectorAdd<<<gridSize, blockSize>>>(deviceInput1, deviceInput2, deviceOutput, size);

    // Copy result back to host
    hipMemcpy(hostOutput, deviceOutput, size * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    std::cout << "Input Array 1: ";
    for (int i = 0; i < size; ++i) std::cout << hostInput1[i] << " ";
    std::cout << "\nInput Array 2: ";
    for (int i = 0; i < size; ++i) std::cout << hostInput2[i] << " ";
    std::cout << "\nOutput Array (Sum): ";
    for (int i = 0; i < size; ++i) std::cout << hostOutput[i] << " ";
    std::cout << std::endl;

    // Free device memory
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    return 0;
}
